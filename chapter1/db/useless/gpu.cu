#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "gpu.h"

__global__ 
void vecAddKernel(float* A_d, float* B_d, float* C_d, int n)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < n) C_d[i] = A_d[i] + B_d[i];
}

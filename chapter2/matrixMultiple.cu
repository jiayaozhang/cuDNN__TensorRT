#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#include "hip/hip_runtime.h"
#include ""
#include "hipblas.h"

#define M 512
#define K 512
#define N 512

#define BLOCK_SIZE 32  //block size ,each thread to calucate each bloc

void initial(float *array, int size)
{
	for (int i = 0; i < size; i++)
	{
		array[i] = (float)(rand() % 10 + 1);
	}
}

void printMatrix(float *array, int row, int col)
{
	float *p = array;
	for (int y = 0; y < row; y++)
	{
		for (int x = 0; x < col; x++)
		{
			printf("%10lf", p[x]);
		}
		p = p + col;
		printf("\n");
	}
	return;
}


void  multiplicateMatrixOnHost(float *array_A, float *array_B, float *array_C, int M_p, int K_p, int N_p)
{
	for (int i = 0; i < M_p; i++)
	{
		for (int j = 0; j < N_p; j++)
		{
			float sum = 0;
			for (int k = 0; k < K_p; k++)
			{
				sum += array_A[i*K_p + k] * array_B[k*N_p + j];
			}
			array_C[i*N_p + j] = sum;
		}
	}

}

__global__ void multiplicateMatrixOnDevice(float *array_A, float *array_B, float *array_C, int M_p, int K_p, int N_p)
{
	int ix = threadIdx.x + blockDim.x*blockIdx.x;//row number
	int iy = threadIdx.y + blockDim.y*blockIdx.y;//col number

	if (ix < N_p && iy < M_p)
	{
		float sum = 0;
		for (int k = 0; k < K_p; k++)
		{
			sum += array_A[iy*K_p + k] * array_B[k*N_p + ix];
		}
		array_C[iy*N_p + ix] = sum;
	}
}

// Compute C = A * B
__global__ void matrixMultiplyShared(float *A, float *B, float *C,
	int numARows, int numAColumns, int numBRows, int numBColumns, int numCRows, int numCColumns)
{
	//@@ Insert code to implement matrix multiplication here
	//@@ You have to use shared memory for this MP

	__shared__ float sharedM[BLOCK_SIZE][BLOCK_SIZE];
	__shared__ float sharedN[BLOCK_SIZE][BLOCK_SIZE];

	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;


	int row = by * BLOCK_SIZE + ty;
	int col = bx * BLOCK_SIZE + tx;


	float Csub = 0.0;

	for (int i = 0; i < (int)(ceil((float)numAColumns / BLOCK_SIZE)); i++)
	{
		//printf("block.x=%d,block.y=%d,threadIdx.x=%d,threadIdx.y=%d,row=%d,col=%d,sharedM[%d][%d]=A[%d],A的值：%f,sharedN[%d][%d]=B[%d],B的值：%f\n",
		//	blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, row, col,
		//	threadIdx.y, threadIdx.x, row*numAColumns + i * BLOCK_SIZE + tx, A[row*numAColumns + i * BLOCK_SIZE + tx],
		//	threadIdx.y, threadIdx.x, (i*BLOCK_SIZE + ty)*numBColumns + col, B[(i*BLOCK_SIZE + ty)*numBColumns + col]);

		if (i*BLOCK_SIZE + tx < numAColumns && row < numARows)
			sharedM[ty][tx] = A[row*numAColumns + i * BLOCK_SIZE + tx];
		else
			sharedM[ty][tx] = 0.0;

		if (i*BLOCK_SIZE + ty < numBRows && col < numBColumns)
			sharedN[ty][tx] = B[(i*BLOCK_SIZE + ty)*numBColumns + col];
		else
			sharedN[ty][tx] = 0.0;
		__syncthreads();


		for (int j = 0; j < BLOCK_SIZE; j++)
			Csub += sharedM[ty][j] * sharedN[j][tx];
		__syncthreads();
	}


	if (row < numCRows && col < numCColumns)
		C[row*numCColumns + col] = Csub;

}


int main(int argc, char **argv)
{
	clock_t start = 0, finish = 0;
	float time;

	int Axy = M * K;
	int Bxy = K * N;
	int Cxy = M * N;


	float *h_A, *h_B, *hostRef, *deviceRef;
	h_A = (float*)malloc(Axy * sizeof(float));
	h_B = (float*)malloc(Bxy * sizeof(float));

	int nBytes = M * N * sizeof(float);
	hostRef = (float*)malloc(Cxy * sizeof(float));
	deviceRef = (float*)malloc(Cxy * sizeof(float));

	initial(h_A, Axy);
	//printf("\n");
	//printf("Matrix_A: (%d×%d)\n", M, K);
	//printMatrix(h_A, M, K);
	initial(h_B, Bxy);
	//printf("Matrix_B: (%d×%d)\n", K, N);
	//printMatrix(h_B, K, N);

	start = clock();
	multiplicateMatrixOnHost(h_A, h_B, hostRef, M, K, N);
	finish = clock();
	time = (float)(finish - start) / CLOCKS_PER_SEC;

	printf("\n");
	printf("------------------------------------------------------------------------------------\n");
	printf("Computing matrix product using multiplicateMatrixOnHost \n");
	printf("------------------------------------------------------------------------------------\n");

	printf("Matrix_hostRef: (%d×%d)  CPU运行时间为：%lfs\n", M, N, time);
	//printMatrix(hostRef, M, N);

	float *d_A, *d_B, *d_C;
	hipMalloc((void**)&d_A, Axy * sizeof(float));
	hipMalloc((void**)&d_B, Bxy * sizeof(float));
	hipMalloc((void**)&d_C, Cxy * sizeof(float));

	hipMemcpy(d_A, h_A, Axy * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, Bxy * sizeof(float), hipMemcpyHostToDevice);


	printf("\n\n");
	printf("------------------------------------------------------------------------------------\n");
	printf("Computing matrix product using multiplicateMatrixOnDevice \n");
	printf("------------------------------------------------------------------------------------\n");

        int dimx = 2;
        int dimy = 2;
        dim3 block(dimx, dimy);
        dim3 grid((M + block.x - 1) / block.x, (N + block.y - 1) / block.y);
        //      dim3 grid(1, 1);

        hipEvent_t gpustart, gpustop;
        float elapsedTime = 0.0;
        hipEventCreate(&gpustart);
        hipEventCreate(&gpustop);
        hipEventRecord(gpustart, 0);
        multiplicateMatrixOnDevice<<<grid,block>>> (d_A, d_B, d_C, M, K, N);
        hipDeviceSynchronize();
        hipEventRecord(gpustop, 0);
        hipEventSynchronize(gpustop);

        hipEventElapsedTime(&elapsedTime, gpustart, gpustop);
        hipEventDestroy(gpustart);
        hipEventDestroy(gpustop);


        hipMemcpy(deviceRef, d_C, Cxy * sizeof(float), hipMemcpyDeviceToHost);
        printf("Matrix_deviceRef: (%d×%d)  <<<(%d,%d),(%d,%d)>>>  GPU运行时间为：%fs\n",
                M, N, grid.x, grid.y, block.x, block.y, elapsedTime / 1000);
        //printMatrix(deviceRef, M, N);


	elapsedTime = 0.0;
	hipEventCreate(&gpustart);
	hipEventCreate(&gpustop);
	hipEventRecord(gpustart, 0);
	matrixMultiplyShared << < grid, block >> > (d_A, d_B, d_C, M, K, K, N, M, N);
	//	printf("   multiplicateMatrixOnDevice<<<(%d,%d),(%d,%d)>>>", grid.x, grid.y, block.x, block.y);
	hipDeviceSynchronize();
	hipEventRecord(gpustop, 0);
	hipEventSynchronize(gpustop);

	hipEventElapsedTime(&elapsedTime, gpustart, gpustop);
	hipEventDestroy(gpustart);
	hipEventDestroy(gpustop);


	hipMemcpy(deviceRef, d_C, Cxy * sizeof(float), hipMemcpyDeviceToHost);
	printf("Matrix_deviceRef: (%d×%d)  <<<(%d,%d),(%d,%d)>>>  GPU运行时间为：%fs\n",
		M, N, grid.x, grid.y, block.x, block.y, elapsedTime / 1000);
	//printMatrix(deviceRef, M, N);
/*
        elapsedTime = 0.0;
        hipEventCreate(&gpustart);
        hipEventCreate(&gpustop);
        hipEventRecord(gpustart, 0);
*/
        hipblasStatus_t status;
        hipblasHandle_t handle;
        hipblasCreate(&handle);

        elapsedTime = 0.0;
        hipEventCreate(&gpustart);
        hipEventCreate(&gpustop);
        hipEventRecord(gpustart, 0);

        float a = 1, b = 0;
        hipblasSgemm(
          handle,
          HIPBLAS_OP_T,   //矩阵A的属性参数，转置，按行优先
          HIPBLAS_OP_T,   //矩阵B的属性参数，转置，按行优先
          M,          //矩阵A、C的行数
          N,          //矩阵B、C的列数
          K,          //A的列数，B的行数，此处也可为B_ROW,一样的
          &a,             //alpha的值
          d_A,            //左矩阵，为A
          K,          //A的leading dimension，此时选择转置，按行优先，则leading dimension为A的列数
          d_B,            //右矩阵，为B
          N,          //B的leading dimension，此时选择转置，按行优先，则leading dimension为B的列数
          &b,             //beta的值
          d_C,            //结果矩阵C
          M           //C的leading dimension，C矩阵一定按列优先，则leading dimension为C的行数
        );
        hipMemcpy(deviceRef, d_C, Cxy * sizeof(float), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        hipEventRecord(gpustop, 0);
        hipEventSynchronize(gpustop);

        hipEventElapsedTime(&elapsedTime, gpustart, gpustop);
        hipEventDestroy(gpustart);
        hipEventDestroy(gpustop);

        printf("Matrix_deviceRef: (%d×%d)  <<<(%d,%d),(%d,%d)>>>  GPU运行时间为：%fs\n",
                M, N, grid.x, grid.y, block.x, block.y, elapsedTime / 1000);

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	free(h_A);
	free(h_B);
	free(hostRef);
	free(deviceRef);

	hipDeviceReset();

	return (0);
}

